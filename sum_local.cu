#include "hip/hip_runtime.h"
%%writefile sum_local.cu

#include <stdio.h>

// Kernel function to add two arrays element-wise
__global__ void add(int *a, int *b, int *c, int n) {
    // Calculate the global index of the current thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Declare a local variable to hold the result of addition
    int local_result = 0;

    // Perform the addition operation for the corresponding elements
    if (i < n)
        local_result = a[i] + b[i];

    // Store the local result in the output array
    //if (i < n)
        c[i] = local_result;
}

int main() {
    int n = 10;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = n * sizeof(int);

    // Allocate memory for arrays a, b, and c on the host
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Initialize arrays a and b on the host
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate memory for arrays d_a, d_b, and d_c on the device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy arrays a and b from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel function with one thread block and n threads
    add<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy array c from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < n; i++)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
